/**********key使用共享内存*************/
/**********使用T-box***********/
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <cstring>
#include <hip/hip_runtime.h>
#include <iomanip>
#include <time.h>

#define BYTE unsigned char
typedef unsigned long u32;

using namespace std;

class aes_block
{
public:
    BYTE block[16];
};

BYTE AES_Sbox[] =
{   /*0    1    2    3    4    5    6    7    8    9    a    b    c    d    e    f */
    0x63,0x7c,0x77,0x7b,0xf2,0x6b,0x6f,0xc5,0x30,0x01,0x67,0x2b,0xfe,0xd7,0xab,0x76, /*0*/ 
    0xca,0x82,0xc9,0x7d,0xfa,0x59,0x47,0xf0,0xad,0xd4,0xa2,0xaf,0x9c,0xa4,0x72,0xc0, /*1*/
    0xb7,0xfd,0x93,0x26,0x36,0x3f,0xf7,0xcc,0x34,0xa5,0xe5,0xf1,0x71,0xd8,0x31,0x15, /*2*/
    0x04,0xc7,0x23,0xc3,0x18,0x96,0x05,0x9a,0x07,0x12,0x80,0xe2,0xeb,0x27,0xb2,0x75, /*3*/
    0x09,0x83,0x2c,0x1a,0x1b,0x6e,0x5a,0xa0,0x52,0x3b,0xd6,0xb3,0x29,0xe3,0x2f,0x84, /*4*/
    0x53,0xd1,0x00,0xed,0x20,0xfc,0xb1,0x5b,0x6a,0xcb,0xbe,0x39,0x4a,0x4c,0x58,0xcf, /*5*/
    0xd0,0xef,0xaa,0xfb,0x43,0x4d,0x33,0x85,0x45,0xf9,0x02,0x7f,0x50,0x3c,0x9f,0xa8, /*6*/ 
    0x51,0xa3,0x40,0x8f,0x92,0x9d,0x38,0xf5,0xbc,0xb6,0xda,0x21,0x10,0xff,0xf3,0xd2, /*7*/
    0xcd,0x0c,0x13,0xec,0x5f,0x97,0x44,0x17,0xc4,0xa7,0x7e,0x3d,0x64,0x5d,0x19,0x73, /*8*/
    0x60,0x81,0x4f,0xdc,0x22,0x2a,0x90,0x88,0x46,0xee,0xb8,0x14,0xde,0x5e,0x0b,0xdb, /*9*/
    0xe0,0x32,0x3a,0x0a,0x49,0x06,0x24,0x5c,0xc2,0xd3,0xac,0x62,0x91,0x95,0xe4,0x79, /*a*/
    0xe7,0xc8,0x37,0x6d,0x8d,0xd5,0x4e,0xa9,0x6c,0x56,0xf4,0xea,0x65,0x7a,0xae,0x08, /*b*/
    0xba,0x78,0x25,0x2e,0x1c,0xa6,0xb4,0xc6,0xe8,0xdd,0x74,0x1f,0x4b,0xbd,0x8b,0x8a, /*c*/
    0x70,0x3e,0xb5,0x66,0x48,0x03,0xf6,0x0e,0x61,0x35,0x57,0xb9,0x86,0xc1,0x1d,0x9e, /*d*/
    0xe1,0xf8,0x98,0x11,0x69,0xd9,0x8e,0x94,0x9b,0x1e,0x87,0xe9,0xce,0x55,0x28,0xdf, /*e*/
    0x8c,0xa1,0x89,0x0d,0xbf,0xe6,0x42,0x68,0x41,0x99,0x2d,0x0f,0xb0,0x54,0xbb,0x16  /*f*/
};

void printBytes(BYTE b[], int len) {
int i;
for (i=0; i<len; i++)
    printf("%x ", b[i]);
printf("\n");
}


void f1printBytes(BYTE b[], int len, FILE* fp) {
int i;
int shiftTab[16]={0,4,8,12,1,5,9,13,2,6,10,14,3,7,11,15};
for (i=0; i<len; i++)
   fprintf(fp, "%02x ", b[shiftTab[i]]);
fprintf(fp, "\n");
}

int flag=0;
void f2printBytes(BYTE b[], int len, FILE* fp) {
int i;
for (i=0; i<len; i++){
   fprintf(fp, "%c", b[i]);
   if(b[i]=='\n')
        flag++;
   }
}

void f3printBytes(BYTE b[], int len, FILE* fp) {
int i;
for (i=0; i<len; i++){
    if(b[i]=='\0')
        return ;
    fprintf(fp, "%c", b[i]);
    if(b[i]=='\n')
    flag++;
   }
}

void AES_ExpandKey(BYTE key[]) {
    int kl = 16, ks=176, Rcon = 1, i, j;
    BYTE temp[4], temp2[4];

    for(i = kl; i < ks; i += 4) {
        memcpy(temp, &key[i-4], 4);
    if (i % kl == 0) {
        temp2[0] = AES_Sbox[temp[1]] ^ Rcon;
        temp2[1] = AES_Sbox[temp[2]];
        temp2[2] = AES_Sbox[temp[3]];
        temp2[3] = AES_Sbox[temp[0]];
        memcpy(temp, temp2, 4);
        if ((Rcon <<= 1) >= 256)
            Rcon ^= 0x11b;
}
    else if ((kl > 24) && (i % kl == 16)) {
        temp2[0] = AES_Sbox[temp[0]];
        temp2[1] = AES_Sbox[temp[1]];
        temp2[2] = AES_Sbox[temp[2]];
        temp2[3] = AES_Sbox[temp[3]];
        memcpy(temp, temp2, 4);
    }
    for(j = 0; j < 4; j++)
        key[i + j] = key[i + j - kl] ^ temp[j];
    }
}

__device__ u32 Byte2Word(BYTE k1, BYTE k2, BYTE k3, BYTE k4){
    u32 result(0x00000000);  
    u32 temp;  
    temp = k1;  // K1  
    temp <<= 24;  
    result |= temp;  
    temp = k2;  // K2  
    temp <<= 16;  
    result |= temp;  
    temp = k3;  // K3  
    temp <<= 8;  
    result |= temp;  
    temp = k4;  // K4  
    result |= temp;  
    return result;  
}

__device__ void AES_Init(u32 keywords[],BYTE key[], int BlockInvTab[], u32 Te0[], u32 Te1[], u32 Te2[], u32 Te3[]) {
    for(int i=0;i<176;i+=4)
        keywords[i/4]=Byte2Word(key[i],key[i+1],key[i+2],key[i+3]);

    BlockInvTab[0]=0;BlockInvTab[1]=4;BlockInvTab[2]=8;BlockInvTab[3]=12;
    BlockInvTab[4]=1;BlockInvTab[5]=5;BlockInvTab[6]=9;BlockInvTab[7]=13;
    BlockInvTab[8]=2;BlockInvTab[9]=6;BlockInvTab[10]=10;BlockInvTab[11]=14;
    BlockInvTab[12]=3;BlockInvTab[13]=7;BlockInvTab[14]=11;BlockInvTab[15]=15;

    u32 T0[256] = {
        0xc66363a5U, 0xf87c7c84U, 0xee777799U, 0xf67b7b8dU,
        0xfff2f20dU, 0xd66b6bbdU, 0xde6f6fb1U, 0x91c5c554U,
        0x60303050U, 0x02010103U, 0xce6767a9U, 0x562b2b7dU,
        0xe7fefe19U, 0xb5d7d762U, 0x4dababe6U, 0xec76769aU,
        0x8fcaca45U, 0x1f82829dU, 0x89c9c940U, 0xfa7d7d87U,
        0xeffafa15U, 0xb25959ebU, 0x8e4747c9U, 0xfbf0f00bU,
        0x41adadecU, 0xb3d4d467U, 0x5fa2a2fdU, 0x45afafeaU,
        0x239c9cbfU, 0x53a4a4f7U, 0xe4727296U, 0x9bc0c05bU,
        0x75b7b7c2U, 0xe1fdfd1cU, 0x3d9393aeU, 0x4c26266aU,
        0x6c36365aU, 0x7e3f3f41U, 0xf5f7f702U, 0x83cccc4fU,
        0x6834345cU, 0x51a5a5f4U, 0xd1e5e534U, 0xf9f1f108U,
        0xe2717193U, 0xabd8d873U, 0x62313153U, 0x2a15153fU,
        0x0804040cU, 0x95c7c752U, 0x46232365U, 0x9dc3c35eU,
        0x30181828U, 0x379696a1U, 0x0a05050fU, 0x2f9a9ab5U,
        0x0e070709U, 0x24121236U, 0x1b80809bU, 0xdfe2e23dU,
        0xcdebeb26U, 0x4e272769U, 0x7fb2b2cdU, 0xea75759fU,
        0x1209091bU, 0x1d83839eU, 0x582c2c74U, 0x341a1a2eU,
        0x361b1b2dU, 0xdc6e6eb2U, 0xb45a5aeeU, 0x5ba0a0fbU,
        0xa45252f6U, 0x763b3b4dU, 0xb7d6d661U, 0x7db3b3ceU,
        0x5229297bU, 0xdde3e33eU, 0x5e2f2f71U, 0x13848497U,
        0xa65353f5U, 0xb9d1d168U, 0x00000000U, 0xc1eded2cU,
        0x40202060U, 0xe3fcfc1fU, 0x79b1b1c8U, 0xb65b5bedU,
        0xd46a6abeU, 0x8dcbcb46U, 0x67bebed9U, 0x7239394bU,
        0x944a4adeU, 0x984c4cd4U, 0xb05858e8U, 0x85cfcf4aU,
        0xbbd0d06bU, 0xc5efef2aU, 0x4faaaae5U, 0xedfbfb16U,
        0x864343c5U, 0x9a4d4dd7U, 0x66333355U, 0x11858594U,
        0x8a4545cfU, 0xe9f9f910U, 0x04020206U, 0xfe7f7f81U,
        0xa05050f0U, 0x783c3c44U, 0x259f9fbaU, 0x4ba8a8e3U,
        0xa25151f3U, 0x5da3a3feU, 0x804040c0U, 0x058f8f8aU,
        0x3f9292adU, 0x219d9dbcU, 0x70383848U, 0xf1f5f504U,
        0x63bcbcdfU, 0x77b6b6c1U, 0xafdada75U, 0x42212163U,
        0x20101030U, 0xe5ffff1aU, 0xfdf3f30eU, 0xbfd2d26dU,
        0x81cdcd4cU, 0x180c0c14U, 0x26131335U, 0xc3ecec2fU,
        0xbe5f5fe1U, 0x359797a2U, 0x884444ccU, 0x2e171739U,
        0x93c4c457U, 0x55a7a7f2U, 0xfc7e7e82U, 0x7a3d3d47U,
        0xc86464acU, 0xba5d5de7U, 0x3219192bU, 0xe6737395U,
        0xc06060a0U, 0x19818198U, 0x9e4f4fd1U, 0xa3dcdc7fU,
        0x44222266U, 0x542a2a7eU, 0x3b9090abU, 0x0b888883U,
        0x8c4646caU, 0xc7eeee29U, 0x6bb8b8d3U, 0x2814143cU,
        0xa7dede79U, 0xbc5e5ee2U, 0x160b0b1dU, 0xaddbdb76U,
        0xdbe0e03bU, 0x64323256U, 0x743a3a4eU, 0x140a0a1eU,
        0x924949dbU, 0x0c06060aU, 0x4824246cU, 0xb85c5ce4U,
        0x9fc2c25dU, 0xbdd3d36eU, 0x43acacefU, 0xc46262a6U,
        0x399191a8U, 0x319595a4U, 0xd3e4e437U, 0xf279798bU,
        0xd5e7e732U, 0x8bc8c843U, 0x6e373759U, 0xda6d6db7U,
        0x018d8d8cU, 0xb1d5d564U, 0x9c4e4ed2U, 0x49a9a9e0U,
        0xd86c6cb4U, 0xac5656faU, 0xf3f4f407U, 0xcfeaea25U,
        0xca6565afU, 0xf47a7a8eU, 0x47aeaee9U, 0x10080818U,
        0x6fbabad5U, 0xf0787888U, 0x4a25256fU, 0x5c2e2e72U,
        0x381c1c24U, 0x57a6a6f1U, 0x73b4b4c7U, 0x97c6c651U,
        0xcbe8e823U, 0xa1dddd7cU, 0xe874749cU, 0x3e1f1f21U,
        0x964b4bddU, 0x61bdbddcU, 0x0d8b8b86U, 0x0f8a8a85U,
        0xe0707090U, 0x7c3e3e42U, 0x71b5b5c4U, 0xcc6666aaU,
        0x904848d8U, 0x06030305U, 0xf7f6f601U, 0x1c0e0e12U,
        0xc26161a3U, 0x6a35355fU, 0xae5757f9U, 0x69b9b9d0U,
        0x17868691U, 0x99c1c158U, 0x3a1d1d27U, 0x279e9eb9U,
        0xd9e1e138U, 0xebf8f813U, 0x2b9898b3U, 0x22111133U,
        0xd26969bbU, 0xa9d9d970U, 0x078e8e89U, 0x339494a7U,
        0x2d9b9bb6U, 0x3c1e1e22U, 0x15878792U, 0xc9e9e920U,
        0x87cece49U, 0xaa5555ffU, 0x50282878U, 0xa5dfdf7aU,
        0x038c8c8fU, 0x59a1a1f8U, 0x09898980U, 0x1a0d0d17U,
        0x65bfbfdaU, 0xd7e6e631U, 0x844242c6U, 0xd06868b8U,
        0x824141c3U, 0x299999b0U, 0x5a2d2d77U, 0x1e0f0f11U,
        0x7bb0b0cbU, 0xa85454fcU, 0x6dbbbbd6U, 0x2c16163aU,
    };

    u32 T1[256] = {
        0xa5c66363U, 0x84f87c7cU, 0x99ee7777U, 0x8df67b7bU,
        0x0dfff2f2U, 0xbdd66b6bU, 0xb1de6f6fU, 0x5491c5c5U,
        0x50603030U, 0x03020101U, 0xa9ce6767U, 0x7d562b2bU,
        0x19e7fefeU, 0x62b5d7d7U, 0xe64dababU, 0x9aec7676U,
        0x458fcacaU, 0x9d1f8282U, 0x4089c9c9U, 0x87fa7d7dU,
        0x15effafaU, 0xebb25959U, 0xc98e4747U, 0x0bfbf0f0U,
        0xec41adadU, 0x67b3d4d4U, 0xfd5fa2a2U, 0xea45afafU,
        0xbf239c9cU, 0xf753a4a4U, 0x96e47272U, 0x5b9bc0c0U,
        0xc275b7b7U, 0x1ce1fdfdU, 0xae3d9393U, 0x6a4c2626U,
        0x5a6c3636U, 0x417e3f3fU, 0x02f5f7f7U, 0x4f83ccccU,
        0x5c683434U, 0xf451a5a5U, 0x34d1e5e5U, 0x08f9f1f1U,
        0x93e27171U, 0x73abd8d8U, 0x53623131U, 0x3f2a1515U,
        0x0c080404U, 0x5295c7c7U, 0x65462323U, 0x5e9dc3c3U,
        0x28301818U, 0xa1379696U, 0x0f0a0505U, 0xb52f9a9aU,
        0x090e0707U, 0x36241212U, 0x9b1b8080U, 0x3ddfe2e2U,
        0x26cdebebU, 0x694e2727U, 0xcd7fb2b2U, 0x9fea7575U,
        0x1b120909U, 0x9e1d8383U, 0x74582c2cU, 0x2e341a1aU,
        0x2d361b1bU, 0xb2dc6e6eU, 0xeeb45a5aU, 0xfb5ba0a0U,
        0xf6a45252U, 0x4d763b3bU, 0x61b7d6d6U, 0xce7db3b3U,
        0x7b522929U, 0x3edde3e3U, 0x715e2f2fU, 0x97138484U,
        0xf5a65353U, 0x68b9d1d1U, 0x00000000U, 0x2cc1ededU,
        0x60402020U, 0x1fe3fcfcU, 0xc879b1b1U, 0xedb65b5bU,
        0xbed46a6aU, 0x468dcbcbU, 0xd967bebeU, 0x4b723939U,
        0xde944a4aU, 0xd4984c4cU, 0xe8b05858U, 0x4a85cfcfU,
        0x6bbbd0d0U, 0x2ac5efefU, 0xe54faaaaU, 0x16edfbfbU,
        0xc5864343U, 0xd79a4d4dU, 0x55663333U, 0x94118585U,
        0xcf8a4545U, 0x10e9f9f9U, 0x06040202U, 0x81fe7f7fU,
        0xf0a05050U, 0x44783c3cU, 0xba259f9fU, 0xe34ba8a8U,
        0xf3a25151U, 0xfe5da3a3U, 0xc0804040U, 0x8a058f8fU,
        0xad3f9292U, 0xbc219d9dU, 0x48703838U, 0x04f1f5f5U,
        0xdf63bcbcU, 0xc177b6b6U, 0x75afdadaU, 0x63422121U,
        0x30201010U, 0x1ae5ffffU, 0x0efdf3f3U, 0x6dbfd2d2U,
        0x4c81cdcdU, 0x14180c0cU, 0x35261313U, 0x2fc3ececU,
        0xe1be5f5fU, 0xa2359797U, 0xcc884444U, 0x392e1717U,
        0x5793c4c4U, 0xf255a7a7U, 0x82fc7e7eU, 0x477a3d3dU,
        0xacc86464U, 0xe7ba5d5dU, 0x2b321919U, 0x95e67373U,
        0xa0c06060U, 0x98198181U, 0xd19e4f4fU, 0x7fa3dcdcU,
        0x66442222U, 0x7e542a2aU, 0xab3b9090U, 0x830b8888U,
        0xca8c4646U, 0x29c7eeeeU, 0xd36bb8b8U, 0x3c281414U,
        0x79a7dedeU, 0xe2bc5e5eU, 0x1d160b0bU, 0x76addbdbU,
        0x3bdbe0e0U, 0x56643232U, 0x4e743a3aU, 0x1e140a0aU,
        0xdb924949U, 0x0a0c0606U, 0x6c482424U, 0xe4b85c5cU,
        0x5d9fc2c2U, 0x6ebdd3d3U, 0xef43acacU, 0xa6c46262U,
        0xa8399191U, 0xa4319595U, 0x37d3e4e4U, 0x8bf27979U,
        0x32d5e7e7U, 0x438bc8c8U, 0x596e3737U, 0xb7da6d6dU,
        0x8c018d8dU, 0x64b1d5d5U, 0xd29c4e4eU, 0xe049a9a9U,
        0xb4d86c6cU, 0xfaac5656U, 0x07f3f4f4U, 0x25cfeaeaU,
        0xafca6565U, 0x8ef47a7aU, 0xe947aeaeU, 0x18100808U,
        0xd56fbabaU, 0x88f07878U, 0x6f4a2525U, 0x725c2e2eU,
        0x24381c1cU, 0xf157a6a6U, 0xc773b4b4U, 0x5197c6c6U,
        0x23cbe8e8U, 0x7ca1ddddU, 0x9ce87474U, 0x213e1f1fU,
        0xdd964b4bU, 0xdc61bdbdU, 0x860d8b8bU, 0x850f8a8aU,
        0x90e07070U, 0x427c3e3eU, 0xc471b5b5U, 0xaacc6666U,
        0xd8904848U, 0x05060303U, 0x01f7f6f6U, 0x121c0e0eU,
        0xa3c26161U, 0x5f6a3535U, 0xf9ae5757U, 0xd069b9b9U,
        0x91178686U, 0x5899c1c1U, 0x273a1d1dU, 0xb9279e9eU,
        0x38d9e1e1U, 0x13ebf8f8U, 0xb32b9898U, 0x33221111U,
        0xbbd26969U, 0x70a9d9d9U, 0x89078e8eU, 0xa7339494U,
        0xb62d9b9bU, 0x223c1e1eU, 0x92158787U, 0x20c9e9e9U,
        0x4987ceceU, 0xffaa5555U, 0x78502828U, 0x7aa5dfdfU,
        0x8f038c8cU, 0xf859a1a1U, 0x80098989U, 0x171a0d0dU,
        0xda65bfbfU, 0x31d7e6e6U, 0xc6844242U, 0xb8d06868U,
        0xc3824141U, 0xb0299999U, 0x775a2d2dU, 0x111e0f0fU,
        0xcb7bb0b0U, 0xfca85454U, 0xd66dbbbbU, 0x3a2c1616U,
    };

    u32 T2[256] = {
        0x63a5c663U, 0x7c84f87cU, 0x7799ee77U, 0x7b8df67bU,
        0xf20dfff2U, 0x6bbdd66bU, 0x6fb1de6fU, 0xc55491c5U,
        0x30506030U, 0x01030201U, 0x67a9ce67U, 0x2b7d562bU,
        0xfe19e7feU, 0xd762b5d7U, 0xabe64dabU, 0x769aec76U,
        0xca458fcaU, 0x829d1f82U, 0xc94089c9U, 0x7d87fa7dU,
        0xfa15effaU, 0x59ebb259U, 0x47c98e47U, 0xf00bfbf0U,
        0xadec41adU, 0xd467b3d4U, 0xa2fd5fa2U, 0xafea45afU,
        0x9cbf239cU, 0xa4f753a4U, 0x7296e472U, 0xc05b9bc0U,
        0xb7c275b7U, 0xfd1ce1fdU, 0x93ae3d93U, 0x266a4c26U,
        0x365a6c36U, 0x3f417e3fU, 0xf702f5f7U, 0xcc4f83ccU,
        0x345c6834U, 0xa5f451a5U, 0xe534d1e5U, 0xf108f9f1U,
        0x7193e271U, 0xd873abd8U, 0x31536231U, 0x153f2a15U,
        0x040c0804U, 0xc75295c7U, 0x23654623U, 0xc35e9dc3U,
        0x18283018U, 0x96a13796U, 0x050f0a05U, 0x9ab52f9aU,
        0x07090e07U, 0x12362412U, 0x809b1b80U, 0xe23ddfe2U,
        0xeb26cdebU, 0x27694e27U, 0xb2cd7fb2U, 0x759fea75U,
        0x091b1209U, 0x839e1d83U, 0x2c74582cU, 0x1a2e341aU,
        0x1b2d361bU, 0x6eb2dc6eU, 0x5aeeb45aU, 0xa0fb5ba0U,
        0x52f6a452U, 0x3b4d763bU, 0xd661b7d6U, 0xb3ce7db3U,
        0x297b5229U, 0xe33edde3U, 0x2f715e2fU, 0x84971384U,
        0x53f5a653U, 0xd168b9d1U, 0x00000000U, 0xed2cc1edU,
        0x20604020U, 0xfc1fe3fcU, 0xb1c879b1U, 0x5bedb65bU,
        0x6abed46aU, 0xcb468dcbU, 0xbed967beU, 0x394b7239U,
        0x4ade944aU, 0x4cd4984cU, 0x58e8b058U, 0xcf4a85cfU,
        0xd06bbbd0U, 0xef2ac5efU, 0xaae54faaU, 0xfb16edfbU,
        0x43c58643U, 0x4dd79a4dU, 0x33556633U, 0x85941185U,
        0x45cf8a45U, 0xf910e9f9U, 0x02060402U, 0x7f81fe7fU,
        0x50f0a050U, 0x3c44783cU, 0x9fba259fU, 0xa8e34ba8U,
        0x51f3a251U, 0xa3fe5da3U, 0x40c08040U, 0x8f8a058fU,
        0x92ad3f92U, 0x9dbc219dU, 0x38487038U, 0xf504f1f5U,
        0xbcdf63bcU, 0xb6c177b6U, 0xda75afdaU, 0x21634221U,
        0x10302010U, 0xff1ae5ffU, 0xf30efdf3U, 0xd26dbfd2U,
        0xcd4c81cdU, 0x0c14180cU, 0x13352613U, 0xec2fc3ecU,
        0x5fe1be5fU, 0x97a23597U, 0x44cc8844U, 0x17392e17U,
        0xc45793c4U, 0xa7f255a7U, 0x7e82fc7eU, 0x3d477a3dU,
        0x64acc864U, 0x5de7ba5dU, 0x192b3219U, 0x7395e673U,
        0x60a0c060U, 0x81981981U, 0x4fd19e4fU, 0xdc7fa3dcU,
        0x22664422U, 0x2a7e542aU, 0x90ab3b90U, 0x88830b88U,
        0x46ca8c46U, 0xee29c7eeU, 0xb8d36bb8U, 0x143c2814U,
        0xde79a7deU, 0x5ee2bc5eU, 0x0b1d160bU, 0xdb76addbU,
        0xe03bdbe0U, 0x32566432U, 0x3a4e743aU, 0x0a1e140aU,
        0x49db9249U, 0x060a0c06U, 0x246c4824U, 0x5ce4b85cU,
        0xc25d9fc2U, 0xd36ebdd3U, 0xacef43acU, 0x62a6c462U,
        0x91a83991U, 0x95a43195U, 0xe437d3e4U, 0x798bf279U,
        0xe732d5e7U, 0xc8438bc8U, 0x37596e37U, 0x6db7da6dU,
        0x8d8c018dU, 0xd564b1d5U, 0x4ed29c4eU, 0xa9e049a9U,
        0x6cb4d86cU, 0x56faac56U, 0xf407f3f4U, 0xea25cfeaU,
        0x65afca65U, 0x7a8ef47aU, 0xaee947aeU, 0x08181008U,
        0xbad56fbaU, 0x7888f078U, 0x256f4a25U, 0x2e725c2eU,
        0x1c24381cU, 0xa6f157a6U, 0xb4c773b4U, 0xc65197c6U,
        0xe823cbe8U, 0xdd7ca1ddU, 0x749ce874U, 0x1f213e1fU,
        0x4bdd964bU, 0xbddc61bdU, 0x8b860d8bU, 0x8a850f8aU,
        0x7090e070U, 0x3e427c3eU, 0xb5c471b5U, 0x66aacc66U,
        0x48d89048U, 0x03050603U, 0xf601f7f6U, 0x0e121c0eU,
        0x61a3c261U, 0x355f6a35U, 0x57f9ae57U, 0xb9d069b9U,
        0x86911786U, 0xc15899c1U, 0x1d273a1dU, 0x9eb9279eU,
        0xe138d9e1U, 0xf813ebf8U, 0x98b32b98U, 0x11332211U,
        0x69bbd269U, 0xd970a9d9U, 0x8e89078eU, 0x94a73394U,
        0x9bb62d9bU, 0x1e223c1eU, 0x87921587U, 0xe920c9e9U,
        0xce4987ceU, 0x55ffaa55U, 0x28785028U, 0xdf7aa5dfU,
        0x8c8f038cU, 0xa1f859a1U, 0x89800989U, 0x0d171a0dU,
        0xbfda65bfU, 0xe631d7e6U, 0x42c68442U, 0x68b8d068U,
        0x41c38241U, 0x99b02999U, 0x2d775a2dU, 0x0f111e0fU,
        0xb0cb7bb0U, 0x54fca854U, 0xbbd66dbbU, 0x163a2c16U,
    };

    u32 T3[256] = {
        0x6363a5c6U, 0x7c7c84f8U, 0x777799eeU, 0x7b7b8df6U,
        0xf2f20dffU, 0x6b6bbdd6U, 0x6f6fb1deU, 0xc5c55491U,
        0x30305060U, 0x01010302U, 0x6767a9ceU, 0x2b2b7d56U,
        0xfefe19e7U, 0xd7d762b5U, 0xababe64dU, 0x76769aecU,
        0xcaca458fU, 0x82829d1fU, 0xc9c94089U, 0x7d7d87faU,
        0xfafa15efU, 0x5959ebb2U, 0x4747c98eU, 0xf0f00bfbU,
        0xadadec41U, 0xd4d467b3U, 0xa2a2fd5fU, 0xafafea45U,
        0x9c9cbf23U, 0xa4a4f753U, 0x727296e4U, 0xc0c05b9bU,
        0xb7b7c275U, 0xfdfd1ce1U, 0x9393ae3dU, 0x26266a4cU,
        0x36365a6cU, 0x3f3f417eU, 0xf7f702f5U, 0xcccc4f83U,
        0x34345c68U, 0xa5a5f451U, 0xe5e534d1U, 0xf1f108f9U,
        0x717193e2U, 0xd8d873abU, 0x31315362U, 0x15153f2aU,
        0x04040c08U, 0xc7c75295U, 0x23236546U, 0xc3c35e9dU,
        0x18182830U, 0x9696a137U, 0x05050f0aU, 0x9a9ab52fU,
        0x0707090eU, 0x12123624U, 0x80809b1bU, 0xe2e23ddfU,
        0xebeb26cdU, 0x2727694eU, 0xb2b2cd7fU, 0x75759feaU,
        0x09091b12U, 0x83839e1dU, 0x2c2c7458U, 0x1a1a2e34U,
        0x1b1b2d36U, 0x6e6eb2dcU, 0x5a5aeeb4U, 0xa0a0fb5bU,
        0x5252f6a4U, 0x3b3b4d76U, 0xd6d661b7U, 0xb3b3ce7dU,
        0x29297b52U, 0xe3e33eddU, 0x2f2f715eU, 0x84849713U,
        0x5353f5a6U, 0xd1d168b9U, 0x00000000U, 0xeded2cc1U,
        0x20206040U, 0xfcfc1fe3U, 0xb1b1c879U, 0x5b5bedb6U,
        0x6a6abed4U, 0xcbcb468dU, 0xbebed967U, 0x39394b72U,
        0x4a4ade94U, 0x4c4cd498U, 0x5858e8b0U, 0xcfcf4a85U,
        0xd0d06bbbU, 0xefef2ac5U, 0xaaaae54fU, 0xfbfb16edU,
        0x4343c586U, 0x4d4dd79aU, 0x33335566U, 0x85859411U,
        0x4545cf8aU, 0xf9f910e9U, 0x02020604U, 0x7f7f81feU,
        0x5050f0a0U, 0x3c3c4478U, 0x9f9fba25U, 0xa8a8e34bU,
        0x5151f3a2U, 0xa3a3fe5dU, 0x4040c080U, 0x8f8f8a05U,
        0x9292ad3fU, 0x9d9dbc21U, 0x38384870U, 0xf5f504f1U,
        0xbcbcdf63U, 0xb6b6c177U, 0xdada75afU, 0x21216342U,
        0x10103020U, 0xffff1ae5U, 0xf3f30efdU, 0xd2d26dbfU,
        0xcdcd4c81U, 0x0c0c1418U, 0x13133526U, 0xecec2fc3U,
        0x5f5fe1beU, 0x9797a235U, 0x4444cc88U, 0x1717392eU,
        0xc4c45793U, 0xa7a7f255U, 0x7e7e82fcU, 0x3d3d477aU,
        0x6464acc8U, 0x5d5de7baU, 0x19192b32U, 0x737395e6U,
        0x6060a0c0U, 0x81819819U, 0x4f4fd19eU, 0xdcdc7fa3U,
        0x22226644U, 0x2a2a7e54U, 0x9090ab3bU, 0x8888830bU,
        0x4646ca8cU, 0xeeee29c7U, 0xb8b8d36bU, 0x14143c28U,
        0xdede79a7U, 0x5e5ee2bcU, 0x0b0b1d16U, 0xdbdb76adU,
        0xe0e03bdbU, 0x32325664U, 0x3a3a4e74U, 0x0a0a1e14U,
        0x4949db92U, 0x06060a0cU, 0x24246c48U, 0x5c5ce4b8U,
        0xc2c25d9fU, 0xd3d36ebdU, 0xacacef43U, 0x6262a6c4U,
        0x9191a839U, 0x9595a431U, 0xe4e437d3U, 0x79798bf2U,
        0xe7e732d5U, 0xc8c8438bU, 0x3737596eU, 0x6d6db7daU,
        0x8d8d8c01U, 0xd5d564b1U, 0x4e4ed29cU, 0xa9a9e049U,
        0x6c6cb4d8U, 0x5656faacU, 0xf4f407f3U, 0xeaea25cfU,
        0x6565afcaU, 0x7a7a8ef4U, 0xaeaee947U, 0x08081810U,
        0xbabad56fU, 0x787888f0U, 0x25256f4aU, 0x2e2e725cU,
        0x1c1c2438U, 0xa6a6f157U, 0xb4b4c773U, 0xc6c65197U,
        0xe8e823cbU, 0xdddd7ca1U, 0x74749ce8U, 0x1f1f213eU,
        0x4b4bdd96U, 0xbdbddc61U, 0x8b8b860dU, 0x8a8a850fU,
        0x707090e0U, 0x3e3e427cU, 0xb5b5c471U, 0x6666aaccU,
        0x4848d890U, 0x03030506U, 0xf6f601f7U, 0x0e0e121cU,
        0x6161a3c2U, 0x35355f6aU, 0x5757f9aeU, 0xb9b9d069U,
        0x86869117U, 0xc1c15899U, 0x1d1d273aU, 0x9e9eb927U,
        0xe1e138d9U, 0xf8f813ebU, 0x9898b32bU, 0x11113322U,
        0x6969bbd2U, 0xd9d970a9U, 0x8e8e8907U, 0x9494a733U,
        0x9b9bb62dU, 0x1e1e223cU, 0x87879215U, 0xe9e920c9U,
        0xcece4987U, 0x5555ffaaU, 0x28287850U, 0xdfdf7aa5U,
        0x8c8c8f03U, 0xa1a1f859U, 0x89898009U, 0x0d0d171aU,
        0xbfbfda65U, 0xe6e631d7U, 0x4242c684U, 0x6868b8d0U,
        0x4141c382U, 0x9999b029U, 0x2d2d775aU, 0x0f0f111eU,
        0xb0b0cb7bU, 0x5454fca8U, 0xbbbbd66dU, 0x16163a2cU,
    };

    for(int i=0;i<256;i++){
        Te0[i]=T0[i];
        Te1[i]=T1[i];
        Te2[i]=T2[i];
        Te3[i]=T3[i];
    }
}

__global__ void AES_Encrypt(aes_block aes_block_array[], BYTE key[],int block_number) {
    int global_thread_index = blockDim.x*blockIdx.x + threadIdx.x;

    __shared__ u32 keywords[44];
    __shared__ int BlockInvTab[16];
    __shared__ u32 Te0[256],Te1[256],Te2[256],Te3[256];


    int stride=blockDim.x*gridDim.x;
    for(int real_thread=global_thread_index;real_thread < block_number;real_thread+=stride){

        if(threadIdx.x == 0 ){
            AES_Init(keywords,key,BlockInvTab,Te0,Te1,Te2,Te3);
            // printf("1");
        }
        __syncthreads();
        u32 s0,s1,s2,s3,t0,t1,t2,t3;
        BYTE block[16]; //定义一个临时存放加密数据的块

        block[0] = aes_block_array[real_thread].block[0];
        block[1] = aes_block_array[real_thread].block[4];
        block[2] = aes_block_array[real_thread].block[8];
        block[3] = aes_block_array[real_thread].block[12];
        block[4] = aes_block_array[real_thread].block[1];
        block[5] = aes_block_array[real_thread].block[5];
        block[6] = aes_block_array[real_thread].block[9];
        block[7] = aes_block_array[real_thread].block[13];
        block[8] = aes_block_array[real_thread].block[2];
        block[9] = aes_block_array[real_thread].block[6];
        block[10] = aes_block_array[real_thread].block[10];
        block[11] = aes_block_array[real_thread].block[14];
        block[12] = aes_block_array[real_thread].block[3];
        block[13] = aes_block_array[real_thread].block[7];
        block[14] = aes_block_array[real_thread].block[11];
        block[15] = aes_block_array[real_thread].block[15];


        /* 初始addkey*/
        s0=Byte2Word(block[0],block[1],block[2],block[3])^keywords[0];
        s1=Byte2Word(block[4],block[5],block[6],block[7])^keywords[1];
        s2=Byte2Word(block[8],block[9],block[10],block[11])^keywords[2];
        s3=Byte2Word(block[12],block[13],block[14],block[15])^keywords[3];

        /*round 1: */
        t0 = Te0[s0 >> 24] ^ Te1[(s1 >> 16) & 0xff] ^ Te2[(s2 >>  8) & 0xff] ^ Te3[s3 & 0xff]^keywords[4];
        t1 = Te0[s1 >> 24] ^ Te1[(s2 >> 16) & 0xff] ^ Te2[(s3 >>  8) & 0xff] ^ Te3[s0 & 0xff]^keywords[5];
        t2 = Te0[s2 >> 24] ^ Te1[(s3 >> 16) & 0xff] ^ Te2[(s0 >>  8) & 0xff] ^ Te3[s1 & 0xff]^keywords[6];
        t3 = Te0[s3 >> 24] ^ Te1[(s0 >> 16) & 0xff] ^ Te2[(s1 >>  8) & 0xff] ^ Te3[s2 & 0xff]^keywords[7];

        /* round 2: */
        s0 = Te0[t0 >> 24] ^ Te1[(t1 >> 16) & 0xff] ^ Te2[(t2 >>  8) & 0xff] ^ Te3[t3 & 0xff] ^ keywords[ 8];
        s1 = Te0[t1 >> 24] ^ Te1[(t2 >> 16) & 0xff] ^ Te2[(t3 >>  8) & 0xff] ^ Te3[t0 & 0xff] ^ keywords[ 9];
        s2 = Te0[t2 >> 24] ^ Te1[(t3 >> 16) & 0xff] ^ Te2[(t0 >>  8) & 0xff] ^ Te3[t1 & 0xff] ^ keywords[10];
        s3 = Te0[t3 >> 24] ^ Te1[(t0 >> 16) & 0xff] ^ Te2[(t1 >>  8) & 0xff] ^ Te3[t2 & 0xff] ^ keywords[11];
        
        /* round 3: */
        t0 = Te0[s0 >> 24] ^ Te1[(s1 >> 16) & 0xff] ^ Te2[(s2 >>  8) & 0xff] ^ Te3[s3 & 0xff] ^ keywords[12];
        t1 = Te0[s1 >> 24] ^ Te1[(s2 >> 16) & 0xff] ^ Te2[(s3 >>  8) & 0xff] ^ Te3[s0 & 0xff] ^ keywords[13];
        t2 = Te0[s2 >> 24] ^ Te1[(s3 >> 16) & 0xff] ^ Te2[(s0 >>  8) & 0xff] ^ Te3[s1 & 0xff] ^ keywords[14];
        t3 = Te0[s3 >> 24] ^ Te1[(s0 >> 16) & 0xff] ^ Te2[(s1 >>  8) & 0xff] ^ Te3[s2 & 0xff] ^ keywords[15];

        /* round 4: */
        s0 = Te0[t0 >> 24] ^ Te1[(t1 >> 16) & 0xff] ^ Te2[(t2 >>  8) & 0xff] ^ Te3[t3 & 0xff] ^ keywords[16];
        s1 = Te0[t1 >> 24] ^ Te1[(t2 >> 16) & 0xff] ^ Te2[(t3 >>  8) & 0xff] ^ Te3[t0 & 0xff] ^ keywords[17];
        s2 = Te0[t2 >> 24] ^ Te1[(t3 >> 16) & 0xff] ^ Te2[(t0 >>  8) & 0xff] ^ Te3[t1 & 0xff] ^ keywords[18];
        s3 = Te0[t3 >> 24] ^ Te1[(t0 >> 16) & 0xff] ^ Te2[(t1 >>  8) & 0xff] ^ Te3[t2 & 0xff] ^ keywords[19];

        /* round 5: */
        t0 = Te0[s0 >> 24] ^ Te1[(s1 >> 16) & 0xff] ^ Te2[(s2 >>  8) & 0xff] ^ Te3[s3 & 0xff] ^ keywords[20];
        t1 = Te0[s1 >> 24] ^ Te1[(s2 >> 16) & 0xff] ^ Te2[(s3 >>  8) & 0xff] ^ Te3[s0 & 0xff] ^ keywords[21];
        t2 = Te0[s2 >> 24] ^ Te1[(s3 >> 16) & 0xff] ^ Te2[(s0 >>  8) & 0xff] ^ Te3[s1 & 0xff] ^ keywords[22];
        t3 = Te0[s3 >> 24] ^ Te1[(s0 >> 16) & 0xff] ^ Te2[(s1 >>  8) & 0xff] ^ Te3[s2 & 0xff] ^ keywords[23];

        /* round 6: */
        s0 = Te0[t0 >> 24] ^ Te1[(t1 >> 16) & 0xff] ^ Te2[(t2 >>  8) & 0xff] ^ Te3[t3 & 0xff] ^ keywords[24];
        s1 = Te0[t1 >> 24] ^ Te1[(t2 >> 16) & 0xff] ^ Te2[(t3 >>  8) & 0xff] ^ Te3[t0 & 0xff] ^ keywords[25];
        s2 = Te0[t2 >> 24] ^ Te1[(t3 >> 16) & 0xff] ^ Te2[(t0 >>  8) & 0xff] ^ Te3[t1 & 0xff] ^ keywords[26];
        s3 = Te0[t3 >> 24] ^ Te1[(t0 >> 16) & 0xff] ^ Te2[(t1 >>  8) & 0xff] ^ Te3[t2 & 0xff] ^ keywords[27];

        /* round 7: */
        t0 = Te0[s0 >> 24] ^ Te1[(s1 >> 16) & 0xff] ^ Te2[(s2 >>  8) & 0xff] ^ Te3[s3 & 0xff] ^ keywords[28];
        t1 = Te0[s1 >> 24] ^ Te1[(s2 >> 16) & 0xff] ^ Te2[(s3 >>  8) & 0xff] ^ Te3[s0 & 0xff] ^ keywords[29];
        t2 = Te0[s2 >> 24] ^ Te1[(s3 >> 16) & 0xff] ^ Te2[(s0 >>  8) & 0xff] ^ Te3[s1 & 0xff] ^ keywords[30];
        t3 = Te0[s3 >> 24] ^ Te1[(s0 >> 16) & 0xff] ^ Te2[(s1 >>  8) & 0xff] ^ Te3[s2 & 0xff] ^ keywords[31];

        /* round 8: */
        s0 = Te0[t0 >> 24] ^ Te1[(t1 >> 16) & 0xff] ^ Te2[(t2 >>  8) & 0xff] ^ Te3[t3 & 0xff] ^ keywords[32];
        s1 = Te0[t1 >> 24] ^ Te1[(t2 >> 16) & 0xff] ^ Te2[(t3 >>  8) & 0xff] ^ Te3[t0 & 0xff] ^ keywords[33];
        s2 = Te0[t2 >> 24] ^ Te1[(t3 >> 16) & 0xff] ^ Te2[(t0 >>  8) & 0xff] ^ Te3[t1 & 0xff] ^ keywords[34];
        s3 = Te0[t3 >> 24] ^ Te1[(t0 >> 16) & 0xff] ^ Te2[(t1 >>  8) & 0xff] ^ Te3[t2 & 0xff] ^ keywords[35];

        /* round 9: */
        t0 = Te0[s0 >> 24] ^ Te1[(s1 >> 16) & 0xff] ^ Te2[(s2 >>  8) & 0xff] ^ Te3[s3 & 0xff] ^ keywords[36];
        t1 = Te0[s1 >> 24] ^ Te1[(s2 >> 16) & 0xff] ^ Te2[(s3 >>  8) & 0xff] ^ Te3[s0 & 0xff] ^ keywords[37];
        t2 = Te0[s2 >> 24] ^ Te1[(s3 >> 16) & 0xff] ^ Te2[(s0 >>  8) & 0xff] ^ Te3[s1 & 0xff] ^ keywords[38];
        t3 = Te0[s3 >> 24] ^ Te1[(s0 >> 16) & 0xff] ^ Te2[(s1 >>  8) & 0xff] ^ Te3[s2 & 0xff] ^ keywords[39];

        /* round 10: */
        s0 =(Te2[(t0 >> 24)] & 0xff000000) ^(Te3[(t1 >> 16) & 0xff] & 0x00ff0000) ^(Te0[(t2 >>8) & 0xff] & 0x0000ff00) ^(Te1[(t3) & 0xff] & 0x000000ff) ^keywords[40];
        s1 =(Te2[(t1 >> 24)] & 0xff000000) ^(Te3[(t2 >> 16) & 0xff] & 0x00ff0000) ^(Te0[(t3 >>8) & 0xff] & 0x0000ff00) ^(Te1[(t0) & 0xff] & 0x000000ff) ^keywords[41];
        s2 =(Te2[(t2 >> 24)] & 0xff000000) ^(Te3[(t3 >> 16) & 0xff] & 0x00ff0000) ^(Te0[(t0 >>8) & 0xff] & 0x0000ff00) ^(Te1[(t1) & 0xff] & 0x000000ff) ^keywords[42];
        s3 =(Te2[(t3 >> 24)] & 0xff000000) ^(Te3[(t0 >> 16) & 0xff] & 0x00ff0000) ^(Te0[(t1 >>8) & 0xff] & 0x0000ff00) ^(Te1[(t2) & 0xff] & 0x000000ff) ^keywords[43];
      
        aes_block_array[real_thread].block[0]=(s0>>24)&0xff;
        aes_block_array[real_thread].block[1]=(s1>>24)&0xff;
        aes_block_array[real_thread].block[2]=(s2>>24)&0xff;
        aes_block_array[real_thread].block[3]=(s3>>24)&0xff;
        aes_block_array[real_thread].block[4]=(s0>>16)&0xff;
        aes_block_array[real_thread].block[5]=(s1>>16)&0xff;
        aes_block_array[real_thread].block[6]=(s2>>16)&0xff;
        aes_block_array[real_thread].block[7]=(s3>>16)&0xff;
        aes_block_array[real_thread].block[8]=(s0>>8)&0xff;
        aes_block_array[real_thread].block[9]=(s1>>8)&0xff;
        aes_block_array[real_thread].block[10]=(s2>>8)&0xff;
        aes_block_array[real_thread].block[11]=(s3>>8)&0xff;
        aes_block_array[real_thread].block[12]=s0&0xff;
        aes_block_array[real_thread].block[13]=s1&0xff;
        aes_block_array[real_thread].block[14]=s2&0xff;
        aes_block_array[real_thread].block[15]=s3&0xff;    
    }
}


int main(int argc, char* argv[]) {
    ifstream ifs;
    ifs.open(argv[1], ios::binary);
    if(!ifs){
        cerr<<"错误：无法打开加密文件"<<endl;
        exit(1);
    }
    ifs.seekg(0, ios::end); 
    int infileLength = ifs.tellg();
    infileLength-=1;
    ifs.seekg(0, ios::beg);
    cout<<"输入文件长度为(字节)： "<<infileLength<<endl<<"文件块个数为： "<<infileLength/16<<endl;

    int block_number = infileLength/16 ;
    int number_of_zero_pending = infileLength%16;
    aes_block* aes_block_array;

    BYTE key[16 * 11]; //定义AES中需要的最大的key
    int keyLen = 0;
    int blockLen = 16;
    ifstream key_fp;
    key_fp.open(argv[2]);
    while(key_fp.peek()!=EOF)
    {
            key_fp>>key[keyLen];
            if(key_fp.eof())
                break;
            keyLen++;
    }

    cout<<"密码长度为（字节）:"<<keyLen<<endl;

    switch (keyLen) 
    {
    case 16:break;
    case 24:break;
    case 32:break;
    default:printf("错误：密钥需要128, 192或256字节\n"); return 0;
    }
    
    AES_ExpandKey(key);

    if(number_of_zero_pending != 0)
        aes_block_array = new aes_block [ block_number + 1];
    else
        aes_block_array = new aes_block[ block_number ];
    char temp[16];

    FILE* en_fp; //定义加密文件
    en_fp = fopen(argv[3], "wb");
    int shiftTab[16]={0,4,8,12,1,5,9,13,2,6,10,14,3,7,11,15};
    for(int i=0; i<block_number; i++){
        ifs.read(temp, 16);
        for(int j=0; j<16; j++){
            aes_block_array[i].block[shiftTab[j]] = (unsigned char)temp[j];
        }
    }
    if(number_of_zero_pending != 0)
    {
        ifs.read(temp, number_of_zero_pending);
        for(int j=0; j<16; j++){
            aes_block_array[block_number].block[j] = (unsigned char)temp[j];
        }
        for(int j=1; j<=16-number_of_zero_pending; j++)
            aes_block_array[block_number].block[16-j] = '\0';
        block_number++;
    }
            
    hipSetDevice(0);	//选择设备
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    int num_sm = prop.multiProcessorCount; 

    aes_block *cuda_aes_block_array;
    BYTE *cuda_key;

    // int thrdperblock = block_number/num_sm;
    // if(block_number%num_sm>0)
    //     thrdperblock++;

    // //设备线程快内线程数最多为1024
    // if(thrdperblock>1024){
    //     thrdperblock = 1024;
    //     num_sm = block_number/1024;
    //     if(block_number%1024>0){
    //         num_sm++;
    //     }
    // }

    dim3 ThreadperBlock(512);
    dim3 BlockperGrid(num_sm);
    // cout<<"线程块数："<<num_sm<<endl;
    // cout<<"块内线程数"<<thrdperblock<<endl;


    hipMalloc(&cuda_aes_block_array, block_number*sizeof(class aes_block));
    hipMalloc(&cuda_key,16*15*sizeof(BYTE) );
    hipMemcpy(cuda_aes_block_array, aes_block_array, block_number*sizeof(class aes_block), hipMemcpyHostToDevice);
    hipMemcpy(cuda_key, key, 16*15*sizeof(BYTE), hipMemcpyHostToDevice);


    printf("加密数据块数: %d\n", block_number);

    hipEvent_t start1;
    hipEventCreate(&start1);
    hipEvent_t stop1;
    hipEventCreate(&stop1);
    hipEventRecord(start1, NULL);
    AES_Encrypt <<< BlockperGrid, ThreadperBlock>>>(cuda_aes_block_array, cuda_key, block_number);
    hipEventRecord(stop1, NULL);
    hipEventSynchronize(stop1);
    float msecTotal1 = 0.0f,total;
    hipEventElapsedTime(&msecTotal1, start1, stop1);
    total=msecTotal1/1000;
    cout<<"加密时间："<<total<<endl;
    long r=1<<23; //单位换算常数
    cout<<"吞吐量为："<<block_number/total/r<<" Gbps"<<endl;

    hipMemcpy(aes_block_array, cuda_aes_block_array, block_number*sizeof(class aes_block), hipMemcpyDeviceToHost);

    for(int i=0; i<block_number; i++)
        f1printBytes(aes_block_array[i].block, blockLen, en_fp);

    return 0;
}
